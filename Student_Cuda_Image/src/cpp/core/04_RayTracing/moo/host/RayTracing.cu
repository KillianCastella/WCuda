#include "hip/hip_runtime.h"
#include "RayTracing.h"

#include <iostream>
#include <assert.h>

#include "SphereCreator.h"

#include "Device.h"
#include "NbSphere.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/
__shared__ Sphere TAB_SM[NB_SPHERE];
extern __global__ void rayTracing(uchar4* ptrDevPixels,uint w, uint h,float t);
extern __global__ void rayTracing(uchar4* ptrDevPixels,uint w, uint h,float t,Sphere* ptrDevTabSphere);
extern void uploadToCM(Sphere* ptrTabSphere);
/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

RayTracing::RayTracing(const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4")
    {
    // Inputs
    this->dt = dt;
    this->nbSphere=NB_SPHERE;

    //memory managament
    SphereCreator sphereCreator(NB_SPHERE, w, h); // sur la pile
    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    this->sizeOctet = NB_SPHERE * sizeof(Sphere); // octet

    // transfert to GM
    Device::malloc(&ptrDevTabSphere, this->sizeOctet);
    Device::memcpyHToD(ptrDevTabSphere, ptrTabSphere, this->sizeOctet);
    //transfert to CM
    uploadToCM(ptrTabSphere);

    // Tools
    this->t = 0; // protected dans Animable
    }

RayTracing::~RayTracing()
    {
    Device::free(ptrDevTabSphere);
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void RayTracing::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("vague rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    rayTracing<<<dg,db>>>(ptrDevPixels,w,h,t); //version CM


    //rayTracing<<<dg,db,sizeOctet>>>(ptrDevPixels,w,h,t,ptrDevTabSphere);//Version SM

    Device::lastCudaError("vague rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void RayTracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

