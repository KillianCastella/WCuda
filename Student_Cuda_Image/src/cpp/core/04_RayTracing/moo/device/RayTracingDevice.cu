#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools_GPU.h"
#include "NbSphere.h"

#include "math/RayTracingMath.h"

using namespace gpu;

__constant__ Sphere TAB_CM[NB_SPHERE];

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    RayTracingMath rayTracingMath = RayTracingMath(w, h, NB_SPHERE, TAB_CM);

    const int WH = w * h;
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID;  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)

	rayTracingMath.colorIJ(&ptrDevPixels[s], i, j, t); 	// update ptrDevPixels[s]

	s += NB_THREAD;
	}
    }
__host__ void uploadToCM(Sphere* ptrTabSphere)
    {
    size_t size = NB_SPHERE * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), ptrTabSphere, size, offset, hipMemcpyHostToDevice));
    }
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

