#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools_GPU.h"
#include "NbSphere.h"

#include "math/RayTracingMath.h"

using namespace gpu;

__constant__ Sphere TAB_CM[NB_SPHERE];

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t);
__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere);
/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__device__ void copyGMtoSM(Sphere* ptrTabSM, Sphere* ptrTabGM, int n)
    {
    const int TID_LOCAL = Indice2D::tidLocal();
    const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();
    int s = TID_LOCAL;
    while (s < n)
	{
	ptrTabSM[s] = ptrTabGM[s];
	s += NB_THREAD_LOCAL;
	}
    }
__device__ void work(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere)
    {
    RayTracingMath rayTracingMath = RayTracingMath(w, h, NB_SPHERE, ptrDevTabSphere);
    const int WH = w * h;
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID;  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)

	rayTracingMath.colorIJ(&ptrDevPixels[s], i, j, t); 	// update ptrDevPixels[s]

	s += NB_THREAD;
	}
    }
__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t)
    {
    work(ptrDevPixels, w, h, t, TAB_CM);
    }
__global__ void rayTracing(uchar4* ptrDevPixels, uint w, uint h, float t, Sphere* ptrDevTabSphere)
    {
    __shared__  extern Sphere
    TAB_SM[];
    copyGMtoSM(TAB_SM, ptrDevTabSphere, NB_SPHERE);

    __syncthreads();
    work(ptrDevPixels, w, h, t, TAB_SM);

    }
__host__ void uploadToCM(Sphere* ptrTabSphere)
    {
    size_t size = NB_SPHERE * sizeof(Sphere);
    int offset = 0;
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TAB_CM), ptrTabSphere, size, offset, hipMemcpyHostToDevice));
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

