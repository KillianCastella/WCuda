#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <iostream>
#include <assert.h>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels,uint w, uint h,float t,DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h,int n, const DomaineMath& domaineMath, float dt) :
	Animable_I<uchar4>(grid, w, h, "Mandelbrot_Cuda_RGBA_uchar4",domaineMath), variateurAnimation(Interval<int>(30,100),dt)
    {
	//inputs
	this->n=n;
	// Tools
	this->t = 0; // protected dans Animable
    }

Mandelbrot::~Mandelbrot()
    {
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("vague rgba uchar4 (before)"); // facultatif, for debug only, remove for release

    mandelbrot<<<dg,db>>>(ptrDevPixels,w,h,t, domaineMath);

    Device::lastCudaError("vague rgba uchar4 (after)"); // facultatif, for debug only, remove for release
    }

/**
 * Override
 * Call periodicly by the API
 */
void Mandelbrot::animationStep()
    {
    t = this->variateurAnimation.varierAndGet();
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

