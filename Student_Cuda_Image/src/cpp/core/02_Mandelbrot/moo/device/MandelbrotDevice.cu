#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"
#include "IndiceTools_GPU.h"
#include "DomaineMath_GPU.h"

#include "math/MandelbrotMath.h"

using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t,int n, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, float t, DomaineMath domaineMath)
    {
    MandelbrotMath mandelbrotMath = MandelbrotMath();

    const int WH = w * h;
    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();

    int i;	// in [0,h[
    int j; 	// in [0,w[

    int s = TID;  // in [0,...
    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); 	// update (i, j)
	float x,y;
	domaineMath.toXY(i, j, &x, &y);

	mandelbrotMath.colorXY(&ptrDevPixels[s], x, y, t); 	// update ptrDevPixels[s]

	s += NB_THREAD;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

