#include "hip/hip_runtime.h"
#include "Slice_advanced.h"

#include <iostream>

#include "Device.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void slice_advanced(float* ptrTabDev,int n);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Constructeur			*|
 \*-------------------------------------*/

Slice_advanced::Slice_advanced(const Grid& grid, int n) :
	n(n)
    {
    // Grid
	{
	this->dg = grid.dg;
	this->db = grid.db;
	}
this->nbThread = grid.threadCounts();
this->sizeOctet = nbThread * sizeof(float); // octet
    // MM
	{

	// MM (malloc Device)
	    {
	    Device::malloc(&ptrTabDev, sizeOctet);
	    }
	ptrTab= new float[nbThread];
	Device::lastCudaError("AddVector MM (end allocation)"); // temp debug, facultatif
	}


    }

Slice_advanced::~Slice_advanced(void)
    {
    //MM (device free)
	{
	Device::free(ptrTabDev);

	Device::lastCudaError("AddVector MM (end deallocation)"); // temp debug, facultatif
	}
    }

/*--------------------------------------*\
 |*		Methode			*|
 \*-------------------------------------*/

void Slice_advanced::run()
    {
    Device::lastCudaError("slice(before)"); // temp debug
    slice_advanced<<<dg,db>>>(ptrTabDev, n); // assynchrone
    Device::lastCudaError("slice (after)"); // temp debug

    //Device::synchronize(); // Temp,debug, only for printf in  GPU

    // MM (Device -> Host)
	{
	Device::memcpyDToH(ptrTab, ptrTabDev, sizeOctet); // barriere synchronisation implicite
	}
	double pi=0;
	for(int i=0;i<nbThread;i++){
	    pi += ptrTab[i];
	}
	pi = pi / (double) n;
	std::cout<<pi;
	//TODO OpenMp reduction
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/
